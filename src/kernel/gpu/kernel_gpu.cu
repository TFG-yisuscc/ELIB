#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <vector>

#define ENABLE_GPU 1

#include "core/tensor.h"
#include "kern/kernel.h"
#include "kernel_gpu.h"
#include "math.h"
#include "string.h"
#include "utils/utils.h"

namespace ELIB {
namespace gpu {

#define CUDA_KERNEL_LOOP(i, n)                                   \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
         i += blockDim.x * gridDim.x)

constexpr int kBlockSize = 256;
constexpr int kNumWaves = 32;
constexpr int DequantizedBlockSize = 256;
const int CUDA_NUM_THREADS = 512;

inline int GET_BLOCKS(const int N) {
    return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

__global__ void llm_elemwise_broadcast_dim0_src1_compute_float_add_gpu(
        const float* src0, const float* src1, float* dst, uint32_t len0,
        uint32_t len1) {
    int row = blockIdx.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (col < len1) {
        int index = row * len1 + col;
        dst[index] = src0[index] + src1[col];
    }
}

__global__ void llm_elemwise_broadcast_dim0_src1_compute_float_mul_gpu(
        const float* src0, const float* src1, float* dst, uint32_t rows,
        uint32_t ncols) {
    int row = blockIdx.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (col < ncols) {
        int index = row * ncols + col;
        dst[index] = src0[index] * src1[col];
    }
}

void llm_elemwise_broadcast_dim0_src1_compute_float(
        const float* src0, const float* src1, float* dst, uint32_t rows, uint32_t ncols,
        ElemMode mode, cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    const dim3 block_dims(512, 1, 1);
    const dim3 block_nums((ncols + 511) / 512, rows, 1);
    switch (mode) {
        case ElemMode::Add: {
            llm_elemwise_broadcast_dim0_src1_compute_float_add_gpu<<<
                    block_nums, block_dims, 0, stream>>>(src0, src1, dst, rows, ncols);
            break;
        }
        case ElemMode::Mul: {
            llm_elemwise_broadcast_dim0_src1_compute_float_mul_gpu<<<
                    block_nums, block_dims, 0, stream>>>(src0, src1, dst, rows, ncols);
            break;
        }
        default:
            INFER_ASSERT(0, "Not supported.");
    }
}


__global__ void softmax_f32_cuda(const float* x, float* dst, const int cols) {
    const int row = blockDim.y * blockIdx.y + threadIdx.y;
    const int block_size = blockDim.x;
    const int tid = threadIdx.x;
    const float* src = x + row * cols;
    dst = dst + row * cols;

    float max = -INFINITY;
    for (int col = tid; col < cols; col += block_size) {
        const float val = src[col];
        max = val > max ? val : max;
    }

    // sum up partial sums
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        float temp = __shfl_xor_sync(0xffffffff, max, mask);
        max = max > temp ? max : temp;
    }

    float sum = 0.0;
    for (int col = tid; col < cols; col += block_size) {
        const float val = expf(src[col] - max);
        sum += val;
        dst[col] = val;
    }

    // sum up partial sums
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        sum += __shfl_xor_sync(0xffffffff, sum, mask, 32);
    }

    for (int col = tid; col < cols; col += block_size) {
        dst[col] /= sum;
    }
}

void llm_softmax_compute_float(
        const float* src, float* dst, uint32_t len_row, uint32_t col,
        cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    const dim3 block_dims(kNumWaves, 1, 1);
    const dim3 block_nums(1, len_row, 1);
    softmax_f32_cuda<<<block_nums, block_dims, 0, stream>>>(src, dst, col);
}

__global__ void embeding_float_cuda(
        const float* weights, const uint32_t* index, float* dst, uint32_t len_seq,
        uint32_t embd) {
    int seq_id = blockIdx.y;
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (thread_id < embd) {
        uint32_t row = index[seq_id];
        dst = dst + seq_id * embd;
        weights = weights + row * embd;
        dst[thread_id] = weights[thread_id];
    }
}

void llm_embedding_get_float_float(
        const float* weights, const uint32_t* index, float* dst, uint32_t len_seq,
        uint32_t embd, cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    const dim3 block_dims(512, 1, 1);
    const dim3 block_nums((embd + 512) / 512, len_seq, 1);
    embeding_float_cuda<<<block_nums, block_dims, 0, stream>>>(
            weights, index, dst, len_seq, embd);
}

__global__ void llm_embedding_get_int4_float_gpu(
        const void* weights, const uint32_t* index, float* dst, uint32_t len_seq,
        uint32_t embd, const int weight_stride) {
    int seq_id = blockIdx.y;
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (thread_id < embd / 2) {
        uint32_t row = index[seq_id];
        dst = dst + seq_id * embd;
        const void* src = (static_cast<const char*>(weights) + row * weight_stride);
        int q40_block_id = thread_id * 2 / QK40;
        int block_offset = thread_id % (QK40 / 2);
        BlockQ40* q40_block = (BlockQ40*)src + q40_block_id;
        float scale = q40_block->d;
        uint8_t value = q40_block->qs[block_offset];
        const int8_t v1 = value & 0xf;
        const int8_t v2 = value >> 4;
        dst[thread_id * 2] = (v1 - 8) * scale;
        dst[thread_id * 2 + 1] = (v2 - 8) * scale;
    }
}

void llm_embedding_get_int4_float(
        const void* weights, const uint32_t* index, float* dst, uint32_t len_seq,
        uint32_t embd, cudaHandle* handle) {
    const int weight_stride = embd * sizeof(BlockQ40) / QK40;
    // one thread compute two data
    int grid_1 = (embd / 2 + DequantizedBlockSize - 1) / DequantizedBlockSize;
    dim3 grid(grid_1, len_seq);
    hipStream_t stream = handle->stream;
    llm_embedding_get_int4_float_gpu<<<grid, DequantizedBlockSize, 0, stream>>>(
            weights, index, dst, len_seq, embd, weight_stride);
}

struct SiluFunctor {
    __device__ float operator()(uint32_t i, const float* input) const {
        float src = input[i];
        return src / (1.0 + exp(-src));
    }
};

struct GeluFunctor {
    __device__ float operator()(uint32_t i, const float* input) const {
        float src = input[i];
        return 0.5 * src * (1 + tanh(sqrt(2.0 / PI) * (src + PGELU * src * src * src)));
    }
};

struct AddFunctor {
    __device__ float operator()(
            uint32_t i, const float* input1, const float* input2) const {
        return input1[i] + input2[i];
    }
};

struct MulFunctor {
    __device__ float operator()(
            uint32_t i, const float* input1, const float* input2) const {
        return input1[i] * input2[i];
    }
};

template <typename Function, typename... Args>
__global__ void ApplyFunction(Function functor, int64_t n, float* ret, Args... args) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        ret[tid] = functor(tid, args...);
    }
}

template <typename Function, typename... Args>
hipError_t LaunchKernel(
        Function fun, hipStream_t stream, int64_t n, float* ret, Args... args) {
    int num_blocks = (n + kBlockSize - 1) / kBlockSize;
    ApplyFunction<Function, Args...>
            <<<num_blocks, kBlockSize, 0, stream>>>(fun, n, ret, args...);
    return hipPeekAtLastError();
}

void llm_elemwise_compute_float(
        InData<float> srcs, float* dst, size_t len, ElemMode mode, cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    switch (mode) {
        case ElemMode::Add: {
            const float* src0 = srcs[0];
            const float* src1 = srcs[1];
            LaunchKernel(AddFunctor(), stream, len, dst, src0, src1);
            break;
        }
        case ElemMode::Mul: {
            const float* src0 = srcs[0];
            const float* src1 = srcs[1];

            LaunchKernel(MulFunctor(), stream, len, dst, src0, src1);
            break;
        }
        case ElemMode::Silu: {
            const float* src0 = srcs[0];
            LaunchKernel(SiluFunctor(), stream, len, dst, src0);
            break;
        }
        case ElemMode::Gelu: {
            const float* src0 = srcs[0];
            LaunchKernel(GeluFunctor(), stream, len, dst, src0);
            break;
        }
        default:
            INFER_ASSERT(0, "Not supported.");
    }
}

__global__ void rms_norm_f32(const float* x, float* dst, const int ncols, float eps) {
    const int row = blockIdx.x;
    const int tid = threadIdx.x;
    const int WARP_SIZE = blockDim.x;

    float tmp = 0.0f;  // partial sum for thread in warp
    for (int i = 0; i < ncols; i += WARP_SIZE) {
        const int col = i + tid;
        const float xi = x[row * ncols + col];
        tmp += xi * xi;
    }

    // sum up partial sums
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    const float mean = tmp / ncols;
    const float scale = 1.0f / sqrtf(mean + eps);

    for (int i = 0; i < ncols; i += WARP_SIZE) {
        const int col = i + tid;
        dst[row * ncols + col] = scale * x[row * ncols + col];
    }
}

void llm_rms_norm_compute_float(
        const float* src, float* dst, uint32_t seq_len, uint32_t embd, float eps,
        cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    rms_norm_f32<<<seq_len, kNumWaves, 0, stream>>>(
            src, dst,  embd, eps);
}

__global__ void norm_f32(const float* x, float* dst, const int ncols, float eps) {
    const int row = blockIdx.x;
    const int tid = threadIdx.x;
    const int WARP_SIZE = blockDim.x;

    float mean = 0.0f;  // partial sum for thread in warp
    for (int i = 0; i < ncols; i += WARP_SIZE) {
        const int col = i + tid;
        const float xi = x[row * ncols + col];
        mean += xi;
    }
    // sum up partial sums
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        mean += __shfl_xor_sync(0xffffffff, mean, mask, 32);
    }
    mean = mean / ncols;

    float sum = 0.0f;  // partial sum for thread in warp
    for (int i = 0; i < ncols; i += WARP_SIZE) {
        const int col = i + tid;
        const float xi = x[row * ncols + col] - mean;
        sum += xi * xi;
        dst[row * ncols + col] = xi;
    }
    // sum up partial sums
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        sum += __shfl_xor_sync(0xffffffff, sum, mask, 32);
    }
    const float scale = 1.0f / sqrtf(sum / ncols + eps);

    for (int i = 0; i < ncols; i += WARP_SIZE) {
        const int col = i + tid;
        dst[row * ncols + col] = scale * x[row * ncols + col];
    }
}

void llm_norm_compute_float(
        const float* src, float* dst, uint32_t seq_len, uint32_t embd, float eps,
        cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    norm_f32<<<seq_len, kNumWaves, 0, stream>>>(src, dst, embd, eps);
}

template <bool halfmode>
__global__ void rope_compute_float(
        float* dst, const float* src, float theta_scale, uint32_t position_offset,
        uint32_t n_rot, uint32_t seqlen, uint32_t n_head, uint32_t head_embd) {
    const int rot = threadIdx.x;
    const int head = blockIdx.x;
    const int seq = blockIdx.y;

    if (rot >= n_rot / 2 || head >= n_head || seq >= seqlen) {
        return;
    }

    const float theta = (position_offset + seq) * powf(theta_scale, rot);
    const float sin_theta = sinf(theta);
    const float cos_theta = cosf(theta);

    const int offset = seq * n_head * head_embd + head * head_embd;
    if (halfmode) {
        const int half_embd = head_embd / 2;
        const float x0 = src[offset + rot];
        const float x1 = src[offset + rot + half_embd];
        dst[offset + rot] = x0 * cos_theta - x1 * sin_theta;
        dst[offset + rot + half_embd] = x0 * sin_theta + x1 * cos_theta;
    } else {
        const float x0 = src[offset + 2 * rot];
        const float x1 = src[offset + 2 * rot + 1];
        dst[offset + 2 * rot] = x0 * cos_theta - x1 * sin_theta;
        dst[offset + 2 * rot + 1] = x0 * sin_theta + x1 * cos_theta;
    }
}

void llm_rope_compute_float(
        float* dst, const float* src, uint32_t n_past, uint32_t n_rot, RotMode m,
        uint32_t seqlen, uint32_t head, uint32_t head_embd, cudaHandle* handle) {
    hipStream_t stream = handle->stream;

    const float theta_scale = powf(10000.0, -2.0f / n_rot);
    const float position_offset = n_past;

    INFER_ASSERT(n_rot <= 2048, "n_rot is two large.");
    INFER_ASSERT(n_rot % 2 == 0, "n_rot must be even.");

    //! offset to nr_past
    if (m == RotMode::Mode1) {
        src = src + n_past * head_embd * head;
        dst = dst + n_past * head_embd * head;
        seqlen = seqlen - n_past;
    }
    const dim3 block_dims(n_rot / 2, 1, 1);
    const dim3 block_nums(head, seqlen, 1);

    if (m == RotMode::ModelRotHalf) {
        rope_compute_float<true><<<block_nums, block_dims, 0, stream>>>(
                dst, src, theta_scale, position_offset, n_rot, seqlen, head, head_embd);
    } else {
        rope_compute_float<false><<<block_nums, block_dims, 0, stream>>>(
                dst, src, theta_scale, position_offset, n_rot, seqlen, head, head_embd);
    }
}

__global__ void llm_elemwise_compute_float_scale_gpu(
        float* src, float* dst, size_t len, float scale) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < len) {
        dst[index] = src[index] * scale;
    }
}

void llm_elemwise_compute_float_scale(
        float* src, float* dst, size_t len, float scale, cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    const dim3 block_dims(CUDA_NUM_THREADS, 1, 1);
    const dim3 block_nums((len + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS, 1, 1);
    llm_elemwise_compute_float_scale_gpu<<<block_nums, block_dims, 0, stream>>>(
            src, dst, len, scale);
}

void llm_matmul_compute_float_float(
        float* dst, const float* src0, const float* bias, const float* src1, uint32_t M,
        uint32_t N, uint32_t K, void* workspace, uint32_t size, cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    hipblasHandle_t cublas_handle = handle->cublas_handle;
    float alpha = 1.f;
    float beta = 0.f;
    CUBLAS_CHECK(hipblasSetStream(cublas_handle, stream));
    CUBLAS_CHECK(hipblasSgemm(
            cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, K,
            &alpha, src0, K, src1, K, &beta, dst, N));
    if (bias != nullptr) {
        llm_elemwise_broadcast_dim0_src1_compute_float(
                dst, bias, dst, M, N, ElemMode::Add, handle);
    }
}

__global__ void dequantize_mul_mat_vec(
        const void* dx, const float* y, const float* bias, float* dst, const int M,
        const int N, const int K) {
        const int m_id = blockIdx.y;
    const int n_id = blockIdx.x * blockDim.y + threadIdx.y;
    const int tid = threadIdx.x;

    if (m_id >= M || n_id >= N) {
        return;
    }

    const int iter_stride = 2 * 32;
    const int vals_per_iter =
            iter_stride / 32;  // num quantized vals per thread and i iter

    // partial sum for each thread
    float tmp = 0.0f;
    const float* srcy = y + m_id * K;
    dst = dst + m_id * N;
    float bias_val = bias ? bias[n_id] : 0.0f;

    for (int i = 0; i < K; i += iter_stride) {
        const int col = i + vals_per_iter * tid;
        const int ib = (n_id * K + col) / QK40;  // x block index
        const int iqs = (col % QK40) / 2;        // x quant index

#pragma unroll
        for (int j = 0; j < vals_per_iter; j += 2) {
            float2 v;
            const BlockQ40* x = (const BlockQ40*)dx + ib;
            const float d = x->d;
            const int vui = x->qs[iqs];

            v.x = vui & 0xF;
            v.y = vui >> 4;

            v.x = (v.x - 8.0f) * d;
            v.y = (v.y - 8.0f) * d;

            tmp += v.x * srcy[col];
            tmp += v.y * srcy[col + 1];
        }
    }

    // sum up partial sums and write back result
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (tid == 0) {
        dst[n_id] = tmp + bias_val;
    }
}

void llm_matmul_compute_int4_float(
        float* dst, const void* src0, const float* bias, const float* src1, uint32_t M,
        uint32_t N, uint32_t K, void* workspace, uint32_t size, cudaHandle* handle) {
    INFER_ASSERT(K % QK40 == 0, "embd is not the time of QK40.");
    hipStream_t stream = handle->stream;
    const dim3 block_nums(N + 15 / 16, M, 1);
    const dim3 block_dims(32, 16, 1);
    dequantize_mul_mat_vec<<<block_nums, block_dims, 0, stream>>>(
            src0, src1, bias, dst, M, N, K);
}

__global__ void llm_scale_diag_mask_inf_float_gpu(
        const float* src, float* dst, const int past, const int len, const int head_dim,
        float scale) {
    const int head = blockIdx.z;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col >= past + len || row >= len || head >= head_dim)
        return;

    const int row_stride = len + past;
    const int head_stride = len * (len + past);

    src = src + head * head_stride + row * row_stride;
    dst = dst + head * head_stride + row * row_stride;

    dst[col] = (col > past + row) ? -INFINITY : src[col] * scale;
}

void llm_scale_diag_mask_inf_float(
        float* dst, const float* src, float scale, uint32_t past, uint32_t seqlen,
        uint32_t head, cudaHandle* handle) {
    hipStream_t stream = handle->stream;

    constexpr int kBlockSize = 32;
    const int block_y = (seqlen + kBlockSize - 1) / kBlockSize;
    const int block_x = (past + seqlen + kBlockSize - 1) / kBlockSize;
    const dim3 block_dims(kBlockSize, kBlockSize, 1);
    const dim3 block_nums(block_x, block_y, head);

    llm_scale_diag_mask_inf_float_gpu<<<block_nums, block_dims, 0, stream>>>(
            src, dst, past, seqlen, head, scale);
}

__global__ void diag_mask_inf_f32(
        const float* src, float* dst, const int past, const int len,
        const int head_dim) {
    const int head = blockIdx.z;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col >= len || row >= len || head >= head_dim)
        return;

    const int row_stride = len + past;
    const int head_stride = len * (len + past);

    src = src + head * head_stride + row * row_stride + past;
    dst = dst + head * head_stride + row * row_stride + past;
    dst[col] = (col > row) ? -INFINITY : src[col];
}

void llm_diag_mask_inf_float(
        float* dst, const float* src, uint32_t n_past, uint32_t N, uint32_t head,
        cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    constexpr int kBlockSize = 32;
    const int block_n = (N + kBlockSize - 1) / kBlockSize;
    const dim3 block_dims(kBlockSize, kBlockSize, 1);
    const dim3 block_nums(block_n, block_n, head);
    diag_mask_inf_f32<<<block_nums, block_dims, 0, stream>>>(src, dst, n_past, N, head);
}

void llm_permute_compute_float(
        float* dst, const float* src0, uint32_t dim0, uint32_t dim1, uint32_t dim2,
        std::vector<uint32_t> param, cudaHandle* handle) {
    return;
}
/**
 * dst :head *seqlen *(seql)
 */

void llm_matmul_compute_with_head_stride_float(
        float* dst, const float* srck, const float* srcq, uint32_t seqlen,
        uint32_t embd, uint32_t head, uint32_t nr_past, cudaHandle* handle) {
    uint32_t head_embd = embd / head;
    uint32_t M = seqlen;
    uint32_t N = seqlen + nr_past;
    uint32_t K = head_embd;
    hipStream_t stream = handle->stream;
    hipblasHandle_t cublas_handle = handle->cublas_handle;
    float alpha = 1.f;
    float beta = 0.f;
    CUBLAS_CHECK(hipblasSetStream(cublas_handle, stream));
    CUBLAS_CHECK(hipblasSgemmStridedBatched(
            cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, K, &alpha, srck, embd,
            head_embd, srcq, embd, head_embd, &beta, dst, N, M * N, head));
}

void llm_head_batched_matmul_compute_float(
        float* dst, const float* v, const float* qk, uint32_t seqlen, uint32_t embd,
        uint32_t head, uint32_t nr_past, cudaHandle* handle) {
    uint32_t head_embd = embd / head;
    uint32_t M = head_embd;
    uint32_t K = seqlen + nr_past;
    uint32_t N = seqlen;
    hipStream_t stream = handle->stream;
    hipblasHandle_t cublas_handle = handle->cublas_handle;
    float alpha = 1.f;
    float beta = 0.f;

    CUBLAS_CHECK(hipblasSetStream(cublas_handle, stream));
    CUBLAS_CHECK(hipblasSgemmStridedBatched(
            cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, v, embd,
            head_embd, qk, K, K * N, &beta, dst, embd, head_embd, head));
}

__global__ void glm_gmask_inf_f32(
        float* dst, const int past, const int seqlen, const int head) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int head_id = index / seqlen;
    const int row_id = index % seqlen;

    // the laxt row not set -inf
    if (row_id >= seqlen - 1 || head_id >= head)
        return;

    int total_seq = seqlen + past;
    int offset = head_id * seqlen * total_seq + row_id * total_seq + total_seq - 1;
    dst[offset] = -INFINITY;
}

void llm_glm_gmask_inf_float(
        float* dst, uint32_t n_past, uint32_t seqlen, uint32_t head,
        cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    uint32_t count = seqlen * head;
    constexpr int kBlockSize = 32 * 32;
    const int block_n = (count + kBlockSize - 1) / kBlockSize;
    const dim3 block_dims(kBlockSize, 1, 1);
    const dim3 block_nums(block_n, 1, 1);
    glm_gmask_inf_f32<<<block_nums, block_dims, 0, stream>>>(dst, n_past, seqlen, head);
}

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define MAX(a, b) ((a) < (b) ? (b) : (a))

__global__ void glm_rope_compute_float(
        float* dst, const float* src, int32_t n_past, int32_t gmask_positon,
        int32_t seqlen, int32_t head, int32_t embd) {
    const int seq = blockIdx.y;
    const int h = blockIdx.x;
    int p = threadIdx.x;

    if (seq >= seqlen || h >= head || p >= embd / 2)
        return;

    int quart_embd = embd / 4;
    int half_embd = embd / 2;

    int position_id = MIN(seq + n_past, gmask_positon);
    int block_position_id = MAX((n_past + seq) - gmask_positon, 0);

    bool is_second_half = p >= quart_embd;

    position_id = is_second_half ? block_position_id : position_id;

    p = p % quart_embd;
    const double theta = pow(10000.0, ((double)-2 * p) / (half_embd));
    const double cos_theta = cos(position_id * theta);
    const double sin_theta = sin(position_id * theta);

    const float* const src_data =
            src + seq * head * embd + h * embd + p + is_second_half * half_embd;
    float* dst_data =
            dst + seq * head * embd + h * embd + p + is_second_half * half_embd;
    double x0 = src_data[0];
    double x32 = src_data[quart_embd];
    dst_data[0] = x0 * cos_theta - x32 * sin_theta;
    dst_data[quart_embd] = x32 * cos_theta + x0 * sin_theta;
}

void llm_glm_rope_compute_float(
        float* dst, const float* src, uint32_t n_past, uint32_t gmask_positon,
        uint32_t seqlen, uint32_t head, uint32_t embd, cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    int half_embd = embd / 2;
    const dim3 block_dims(half_embd, 1, 1);
    const dim3 block_nums(head, seqlen, 1);

    glm_rope_compute_float<<<block_nums, block_dims, 0, stream>>>(
            dst, src, n_past, gmask_positon, seqlen, head, embd);
}

void llm_matmul_compute_with_head_strideq_broadcastk_float(
        float* dst, const float* srck, const float* srcq, uint32_t seqlen,
        uint32_t embd, uint32_t head, uint32_t query_group_num, uint32_t nr_past,
        cudaHandle* handle) {
    uint32_t head_embd = embd / head;
    uint32_t M = seqlen;
    uint32_t N = seqlen + nr_past;
    uint32_t K = head_embd;
    hipStream_t stream = handle->stream;
    hipblasHandle_t cublas_handle = handle->cublas_handle;
    float alpha = 1.f;
    float beta = 0.f;
    CUBLAS_CHECK(hipblasSetStream(cublas_handle, stream));
    uint32_t kv_head_number = head / query_group_num;
    uint32_t stride_k = query_group_num * head_embd;

    // float cpuk[query_group_num * head_embd * seqlen];
    // float cpuq[head * head_embd * seqlen];
    // float cpudst[head * head_embd * seqlen];

    // hipMemcpy(
    //         cpuk, srck, sizeof(float) * query_group_num * head_embd * seqlen,
    //         hipMemcpyDeviceToHost);
    // hipMemcpy(
    //         cpuq, srcq, sizeof(float) * head * head_embd * seqlen,
    //         hipMemcpyDeviceToHost);

    for (int i = 0; i < query_group_num; i++) {
        const float* srck_group = srck + i * head_embd;
        const float* srcq_group = srcq + i * kv_head_number * head_embd;
        float* dst_group = dst + i * kv_head_number * N * M;

        CUBLAS_CHECK(hipblasSgemmStridedBatched(
                cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, K, &alpha, srck_group,
                stride_k, 0, srcq_group, embd, head_embd, &beta, dst_group, N,
                M * N, kv_head_number));
    }
    // hipMemcpy(
    //         cpudst, dst, sizeof(float) * head * head_embd * seqlen,
    //         hipMemcpyDeviceToHost);
}

void llm_head_batched_matmul_broadcastv_float(
        float* dst, const float* v, const float* qk, uint32_t seqlen, uint32_t embd,
        uint32_t head, uint32_t query_group_num, uint32_t nr_past, cudaHandle* handle) {
    uint32_t head_embd = embd / head;
    uint32_t M = head_embd;
    uint32_t K = seqlen + nr_past;
    uint32_t N = seqlen;
    hipStream_t stream = handle->stream;
    hipblasHandle_t cublas_handle = handle->cublas_handle;
    float alpha = 1.f;
    float beta = 0.f;
    CUBLAS_CHECK(hipblasSetStream(cublas_handle, stream));

    uint32_t kv_head_number = head / query_group_num;
    uint32_t stride_v = query_group_num * head_embd;

    for (int i = 0; i < query_group_num; i++) {
        const float* qk_group = qk + i * kv_head_number * K * N;
        const float* srcv_group = v + i * head_embd;
        float* dst_group = dst + i * kv_head_number * head_embd;

        CUBLAS_CHECK(hipblasSgemmStridedBatched(
                cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, srcv_group,
                stride_v, 0, qk_group, K, K * N, &beta, dst_group, embd, head_embd,
                kv_head_number));
    }
}

size_t llm_matmul_get_workspace_float(uint32_t M, uint32_t N, uint32_t K) {
    return 0;
}

size_t llm_matmul_get_workspace_float_float(uint32_t M, uint32_t N, uint32_t K) {
    return 0;
}

}  // namespace gpu
}  // namespace ELIB
